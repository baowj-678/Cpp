/**
 * Description: 获取CUDA属性
 * Author: Bao Wenjie
 * Date: 2021/3/4
 */

#include <hip/hip_runtime.h>

#include <iostream>

using namespace ::std;


int main()
{
	hipError_t cudaStatus;
	int num = 0;
	cudaStatus = hipGetDeviceCount(&num);
	cout << "共有：" << num << "块 GPU" << endl << endl;
	hipDeviceProp_t prop;
	for (int i = 0; i < num; i++)
	{
		hipGetDeviceProperties(&prop, i);
		cout << "第" << i << "块 GPU" << endl;
		cout << "GPU名称: " << prop.name << endl;
		cout << "显存大小为: " << prop.totalGlobalMem << endl;
		cout << "版本号: " << prop.major << "." << prop.minor << endl;
		cout << "GPU大核数: " << prop.multiProcessorCount << endl;
	}
	
}